#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <ctime>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#include "dxt1.h"
#include "types.h"
#include "colors.h"

int main() {
	char inPath[] = "./images/pretty.png";
	char outPath[] = "./output/out.png";
	int width, height, bytesPerPixel;
	long pixelCount;
	BYTE *imageData;

	BYTE *oldImageData = stbi_load(inPath, &width, &height, &bytesPerPixel, 0);
	if (oldImageData == NULL) {
		printf("Invalid image!");
		return  1;
	}
	if (width % 4 != 0 || height % 4 != 0) {
		printf("Invalid dimensions");
		return  1;
	}
	if (width > 32768 || height > 32768) {
		printf("Max image dimensions: 32768x32768");
		return  1;
	}

	pixelCount = width*height;

	// Check if image is RGB and convert to RGBA
	imageData = (BYTE *)malloc(pixelCount * 4);
	if (bytesPerPixel == 3) {
		bytesPerPixel = 4;
		rgb_to_rgba_image(oldImageData, imageData, pixelCount);
	} else if (bytesPerPixel == 4) {
		memcpy(imageData, oldImageData, pixelCount*bytesPerPixel);
	} else {
		printf("Invalid pixel size! %d %d %d", bytesPerPixel, width, height);
		return 1;
	}

	BYTE *compressedImage = (BYTE*)malloc(pixelCount * 4 / 8);
	BYTE* decompressedImage = (BYTE*)malloc(pixelCount * 4);

	time_t compressStart = clock();
	// Compress image DXT1(8:1 compression ratio)
	CompressImageDXT1(imageData, compressedImage, width, height);
	time_t compressEnd = clock();

	// Decompress image DXT1
	DecompressImageDXT1(width, height, compressedImage, decompressedImage);

	time_t decompressEnd = clock();

	printf("compress: %.3fsec\n", double(compressEnd - compressStart) / CLOCKS_PER_SEC);
	printf("decompress: %.3fsec\n", double(decompressEnd - compressEnd) / CLOCKS_PER_SEC);
	printf("total: %.3fsec\n", double(decompressEnd - compressStart) / CLOCKS_PER_SEC);


	//if (stbi_write_jpg(outPath, width, height, bytesPerPixel, imageData, 100)) {
	if (stbi_write_png(outPath, width, height, bytesPerPixel, decompressedImage, width*bytesPerPixel)) {
		printf("WRITE SUCCESS!");
	} else {
		printf("WRITE ERROR!");
	}

	stbi_image_free(imageData);
	stbi_image_free(oldImageData);

	return 0;
}